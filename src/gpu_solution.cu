#include "hip/hip_runtime.h"
#include "algorithm_choices.h"

#include <chrono>    // for timing
#include <iostream>  // std::cout, std::endl

#include "cuda_common.h"

namespace csc485b {
namespace a1      {
namespace gpu     {

/**
 * The CPU baseline benefits from warm caches because the data was generated on
 * the CPU. Run the data through the GPU once with some arbitrary logic to
 * ensure that the GPU cache is warm too and the comparison is more fair.
 */
__global__
void warm_the_gpu( element_t * data, std::size_t invert_at_pos, std::size_t num_elements )
{
    int const th_id = blockIdx.x * blockDim.x + threadIdx.x;

    // We know this will never be true, because of the data generator logic,
    // but I doubt that the compiler will figure it out. Thus every element
    // should be read, but none of them should be modified.
    if( th_id < num_elements && data[ th_id ] > num_elements * 100 )
    {
        ++data[ th_id ]; // should not be possible.
    }
}

/**
 * Your solution. Should match the CPU output.
 */
__global__
void opposing_sort( element_t * data, std::size_t invert_at_pos, std::size_t num_elements )
{
    int const th_id = blockIdx.x * blockDim.x + threadIdx.x;

    if( th_id < num_elements )
    {
        // IMPLEMENT ME!
        return;
    }
}

/**
 * Performs all the logic of allocating device vectors and copying host/input
 * vectors to the device. Times the opposing_sort() kernel with wall time,
 * but excludes set up and tear down costs such as mallocs, frees, and memcpies.
 */
void run_gpu_soln( std::vector< element_t > data, std::size_t switch_at, std::size_t n )
{
    // Kernel launch configurations. Feel free to change these.
    // This is set to maximise the size of a thread block on a T4, but it hasn't
    // been tuned. It's not known if this is optimal.
    std::size_t const threads_per_block = 1024;
    std::size_t const num_blocks =  ( n + threads_per_block - 1 ) / threads_per_block;

    // Allocate arrays on the device/GPU
    element_t * d_data;
    hipMalloc( (void**) & d_data, sizeof( element_t ) * n );
    CHECK_ERROR("Allocating input array on device");

    // Copy the input from the host to the device/GPU
    hipMemcpy( d_data, data.data(), sizeof( element_t ) * n, hipMemcpyHostToDevice );
    CHECK_ERROR("Copying input array to device");

    // Warm the cache on the GPU for a more fair comparison
    warm_the_gpu<<< num_blocks, threads_per_block>>>( d_data, switch_at, n );

    // Time the execution of the kernel that you implemented
    auto const kernel_start = std::chrono::high_resolution_clock::now();
    opposing_sort<<< num_blocks, threads_per_block>>>( d_data, switch_at, n );
    auto const kernel_end = std::chrono::high_resolution_clock::now();
    CHECK_ERROR("Executing kernel on device");

    // After the timer ends, copy the result back, free the device vector,
    // and echo out the timings and the results.
    hipMemcpy( data.data(), d_data, sizeof( element_t ) * n, hipMemcpyDeviceToHost );
    CHECK_ERROR("Transferring result back to host");
    hipFree( d_data );
    CHECK_ERROR("Freeing device memory");

    std::cout << "GPU Solution time: "
              << std::chrono::duration_cast<std::chrono::nanoseconds>(kernel_end - kernel_start).count()
              << " ns" << std::endl;

    for( auto const x : data ) std::cout << x << " "; std::cout << std::endl;
}

} // namespace gpu
} // namespace a1
} // namespace csc485b